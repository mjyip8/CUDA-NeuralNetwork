#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include "mpi.h"

#define GLOBAL 1
#define SMEM 2
#define STRIDED_SMEM 3
#define BLOCK_SIZE 32

#define SMAX_STRIDE 16
#define N_CLASSES 10
#define SUM_STRIDE 32

#define MPI_SAFE_CALL(call)                                                  \
  do {                                                                       \
    int err = call;                                                          \
    if (err != MPI_SUCCESS) {                                                \
      fprintf(stderr, "MPI error %d in file '%s' at line %i", err, __FILE__, \
              __LINE__);                                                     \
      exit(1);                                                               \
    }                                                                        \
  } while (0)

/***************************************************************
 *                           KERNELS
 ***************************************************************/

/* MatMul Kernels */

__global__
void globalMM(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real alpha, real beta,
           int M, int N, int K, 
           bool isVec, bool transposeA, bool transposeB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        real accumulator = 0;  
        for (int i = 0; i < K; i++) {
            int A_index = (transposeA)? i + row*K : row + i*M;
            int B_index = (transposeB)? col + i*N : i + col*K;
            if (A_index < M * K && B_index < N * K) {
                real a = A[A_index];
                real b = B[B_index];
                accumulator += a * b;
            }
        }
        real add_term = 0.;
        if (beta) {
            real C_term = (isVec) ? C[row + N*M] : C[row + col * M];
            add_term = beta * C_term;
        }
        C[row + col * M] = alpha * accumulator + add_term;
    }
}

__global__
void smeMM(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real alpha, real beta,
           int M, int N, const int K, 
           bool isVec, bool transposeA, bool transposeB) {

    int blockRow = blockIdx.y * blockDim.y;
    int blockCol = blockIdx.x * blockDim.x;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int gRow = blockRow + row;
    int gCol = blockCol + col;

    real accumulator = 0;
    for (int i = 0; i < K; i += BLOCK_SIZE) {
        __syncthreads();
        __shared__ real A_block[BLOCK_SIZE * BLOCK_SIZE];
        __shared__ real B_block[BLOCK_SIZE * BLOCK_SIZE];
        if (gRow < M && i + col < K) {
            int gA_index = (transposeA)? (i + col) + gRow * K : gRow + (i + col) * M;
            A_block[row + col * BLOCK_SIZE] = A[gA_index];
        }
        if (gCol < N && i + row < K) {
            int gB_index = (transposeB)? gCol + (i + row) * N : (i + row) + gCol * K;
            B_block[row + col * BLOCK_SIZE] = B[gB_index];
        }

        __syncthreads();
        
        if (gRow < M && gCol < N) {
            for (int j = 0; j < min(BLOCK_SIZE, K - i); j++) {
                accumulator += A_block[row + j * BLOCK_SIZE] * B_block[j + col * BLOCK_SIZE];
            }
        }
    }

    if (gRow < M && gCol < N) {
        real add_term = 0.;
        if (beta) {
            real C_term = (isVec) ? C[gRow + N*M] : C[gRow + gCol * M];
            add_term = beta * C_term;
        }
        C[gRow + gCol * M] = alpha * accumulator + add_term;
    }
}

// TODO: do transposeA and transposeB tests later
__global__
void stridedSMEMM(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real alpha, real beta,
           int M, int N, int K, 
           bool isVec, bool transposeA, bool transposeB) {
    int blockRow = blockIdx.y * 64;
    int blockCol = blockIdx.x * 16;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int tr = threadIdx.y + blockDim.y * threadIdx.x;
    int gRow = blockRow + tr;

    if (blockRow >= M || blockCol >= N) return;

    real accumulator[16];
    memset(&accumulator[0], 0, sizeof(real) * 16);               
    real A_miniblock[4];

    __shared__ real B_block[16][4];

    for (int ptr = 0; ptr < K; ptr += 4) {
        int br = ptr + col;
        int bc = blockCol + row;
        if (br < K && bc < N)
            B_block[row][col] = B[br + bc * K];

        __syncthreads();

        memset(&A_miniblock[0], 0, sizeof(real) * 4);

        # pragma unroll 
        for (int i = 0; i < 4 && ptr + i < K; ++i) {
            A_miniblock[i] = A[gRow + (ptr + i) * M];
        }

        # pragma unroll 
        for (int c = 0; c < 16; ++c) {
            for (int r = 0; r < 4; ++r) {
                accumulator[c] += A_miniblock[r] * B_block[c][r];
            }
        }

        __syncthreads();

    }

    if (gRow >= M) return;

    # pragma unroll
    for (int c = 0; c < 16; ++c) {
        int idx = gRow + (blockCol + c) * M;
        int add_idx = idx;
        if (blockCol + c >= N) break;
        C[idx] = accumulator[c] * alpha + C[add_idx] * beta;
    }
}

/*------------------ FORWARD PASS KERNELS ---------------------*/

__global__
void sigmoid(real* Z, int M, int N) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;
    const int col = blockDim.y * blockIdx.y + threadIdx.y;
    if (row >= M || col >= N) return;
    Z[row + col * M] = 1. / (1. + exp(-Z[row + col * M]));
}

__global__
void softmax(real* Z, int M, int N) {
    int gCol = blockDim.x * blockIdx.x + threadIdx.x;
    int gRow = threadIdx.y;
    int col = threadIdx.x;
    int row = threadIdx.y;
    int psum_stride = N_CLASSES/2;

    __shared__ real smem_exp[SMAX_STRIDE][N_CLASSES];
    __shared__ real smem_sum[SMAX_STRIDE][2];

    if (gCol < N) {
        smem_exp[col][row] = exp(Z[gRow + gCol * M]);
    
        __syncthreads();

        if (row % psum_stride == 0) {
            int psum_idx = row / psum_stride;
            smem_sum[col][psum_idx] = 0;
            for (int i = 0; i < psum_stride; ++i) {
                smem_sum[col][psum_idx] += smem_exp[col][row + i];
            }
        }
        __syncthreads();

        if (row == 0) {
            smem_sum[col][0] += smem_sum[col][1];
        }
        __syncthreads();

        Z[gRow + gCol * M] = smem_exp[col][row] / smem_sum[col][0];
    }
}

/*------------------ BACKWARD PASS KERNELS ---------------------*/

__global__ 
void subtract(real* A, real*B, real k, int M, int N) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;
    const int col = blockDim.y * blockIdx.y + threadIdx.y;
    if (row >= M || col >= N) return;
    B[row + col * M] = k * (A[row + col * M] - B[row + col * M]);
}

__global__ 
void updateParam(real* A, real*B, real lr, int M, int N) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;
    const int col = blockDim.y * blockIdx.y + threadIdx.y;
    if (row >= M || col >= N) return;
    B[row + col * M] = B[row + col * M] - (lr * A[row + col * M]);
}

__global__ 
void sigmoidBackward(real* S, real* A, int M, int N) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;
    const int col = blockDim.y * blockIdx.y + threadIdx.y;
    if (row >= M || col >= N) return;
    real s = S[row + col * M];
    real a = A[row + col * M];
    A[row + col * M] = a * s * (1. - s);
}

template <unsigned int blockSize>
__device__ void reduce(volatile real *sdata, const int col) {
    if (blockSize >=  64) { sdata[col] += sdata[col + 32]; }
    if (blockSize >=  32) { sdata[col] += sdata[col + 16]; }
    if (blockSize >=  16) { sdata[col] += sdata[col +  8]; }
    if (blockSize >=   8) { sdata[col] += sdata[col +  4]; }
    if (blockSize >=   4) { sdata[col] += sdata[col +  2]; }
    if (blockSize >=   2) { sdata[col] += sdata[col +  1]; }
}

template <unsigned int blockSize>
__global__ void sum(real* __restrict__ A, real* __restrict__ out, real k, int K, int N) {
    extern __shared__ real sdata[];

    const int col = threadIdx.x;
    const int row = blockIdx.x;

    // load into shared memory
    unsigned int i = col;
    sdata[col] = 0;
    while (i < N) { 
        sdata[col] += A[row + i * K] + 
                    ((i + blockSize < N)? A[row + (i + blockSize) * K] : 0); 
        i += (2 * blockSize); 
    }

    __syncthreads();

    if (blockSize >= 512) { if (col < 256) { sdata[col] += sdata[col + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (col < 128) { sdata[col] += sdata[col + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (col <  64) { sdata[col] += sdata[col +  64]; } __syncthreads(); }

    if (col < 32) reduce<blockSize>(sdata, col);

    if (col == 0) out[row] = k * sdata[0];
}

/***************************************************************
 *                           WRAPPERS
 ***************************************************************/

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C

m, n, k
    Integers indicating the size of the matrices:
    A: m rows by k columns
    B: k rows by n columns
    C: m rows by n columns
*/
int myGEMM(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real* alpha, real* beta,
           int M, int N, int K, 
           bool isVec, bool transposeA, bool transposeB) {
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int algorithm = STRIDED_SMEM;
    switch (algorithm) {
        case GLOBAL:
            globalMM<<<blocks, threads>>>(A, B, C, *alpha, *beta, M, N, K, isVec, transposeA, transposeB);
            break;
        case SMEM:
            smeMM<<<blocks, threads>>>(A, B, C, *alpha, *beta, M, N, K, isVec, transposeA, transposeB);
            break;
        case STRIDED_SMEM:
            dim3 stridedThreads(4, 16);
            dim3 stridedBlocks((N + 15) / 16, (M + 63) / 64);
            stridedSMEMM<<<stridedBlocks, stridedThreads>>>(A, B, C, *alpha, *beta, M, N, K, isVec, transposeA, transposeB);
            break;
      }
  return 0;
}

// without pointers to weights
int myGEMM(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real alpha, real beta,
           int M, int N, int K, 
           bool isVec, bool transposeA, bool transposeB) {
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int algorithm = GLOBAL;
    switch (algorithm) {
        case GLOBAL:
            globalMM<<<blocks, threads>>>(A, B, C, alpha, beta, M, N, K, isVec, transposeA, transposeB);
            break;
        case SMEM:
            smeMM<<<blocks, threads>>>(A, B, C, alpha, beta, M, N, K, isVec, transposeA, transposeB);
            break;
      }
  return 0;
}

int myGEMMAsync(real* __restrict__ A, real* __restrict__ B,
           real* __restrict__ C, real alpha, real beta,
           int M, int N, int K, CudaStreams& streams, int i,
           bool isVec, bool transposeA, bool transposeB) {
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int algorithm = GLOBAL;
    switch (algorithm) {
        case GLOBAL:
            globalMM<<<blocks, threads, 0, streams.stream[i]>>>(A, B, C, alpha, beta, M, N, K, isVec, transposeA, transposeB);
            break;
        case SMEM:
            smeMM<<<blocks, threads, 0, streams.stream[i]>>>(A, B, C, alpha, beta, M, N, K, isVec, transposeA, transposeB);
            break;
      }
  return 0;
}

/*------------------ FORWARD PASS WRAPPERS ---------------------*/

void deviceSigmoid(real* Z, int M, int N) {
    real* result;
    checkCudaErrors(hipMalloc(&result, sizeof(real) * M * N));

    dim3 threadDims(32, 32);
    dim3 blockDims((M + threadDims.x - 1) / threadDims.x, (N + threadDims.y - 1) / threadDims.y);
    sigmoid<<<blockDims, threadDims>>>(Z, M, N);
}

void deviceSoftmax(real* Z, int M, int N) {
    dim3 threadDims(SMAX_STRIDE, M);
    dim3 blockDims((N + threadDims.x - 1) / threadDims.x, 1);
    softmax<<<blockDims, threadDims>>>(Z, M, N);
}

/*------------------ BACKWARD PASS WRAPPERS ---------------------*/

void deviceSubtract(real* A, real*B, real k, int M, int N) {
    dim3 threadDims(32, 32);
    dim3 blockDims((M + threadDims.x - 1) / threadDims.x, (N + threadDims.y) / threadDims.y);
    subtract<<<blockDims, threadDims>>>(A, B, k, M, N);
}

void deviceSum(real* A, real* result, real k, int K, int N, CudaStreams& streams, int i) {

    // batch_size cases: 800, 400, 267, 266, 200, 100
    dim3 gridDim(K);
    dim3 blockDim(K);
    int threads;
    size_t sharedMemSize;

    int floor_N = (N / 2) * 2;
    if (floor_N > 512) {
        threads = 512;
        blockDim.x = threads;
        sharedMemSize = threads * sizeof(real);
        sum<512><<<gridDim, blockDim, sharedMemSize, streams.stream[i]>>>(A, result, k, K, N);
    } else if (floor_N <= 512 && floor_N > 256) { 
        threads = 256;
        blockDim.x = threads;
        sharedMemSize = threads * sizeof(real);
        sum<256><<<gridDim, blockDim, sharedMemSize, streams.stream[i]>>>(A, result, k, K, N);
    } else if (floor_N <= 256 && floor_N > 128) {
        threads = 128;
        blockDim.x = threads;
        sharedMemSize = threads * sizeof(real);
        sum<128><<<gridDim, blockDim, sharedMemSize, streams.stream[i]>>>(A, result, k, K, N);
    } else if (floor_N <= 128 && floor_N > 64) {
        threads = 64;
        blockDim.x = threads;
        sharedMemSize = threads * sizeof(real);
        sum<64><<<gridDim, blockDim, sharedMemSize, streams.stream[i]>>>(A, result, k, K, N);
    } else {
        threads = 32;
        blockDim.x = threads;
        sharedMemSize = threads * sizeof(real);
        sum<32><<<gridDim, blockDim, sharedMemSize, streams.stream[i]>>>(A, result, k, K, N);
    }
}

void deviceSigmoidBackward(real* S, real* A, int M, int N) {
    dim3 threadDims(32, 32);
    dim3 blockDims((M + threadDims.x - 1) / threadDims.x, (N + threadDims.y) / threadDims.y);
    sigmoidBackward<<<blockDims, threadDims>>>(S, A, M, N);
}

void deviceUpdateParam(real* A, real*B, real lr, int M, int N, hipStream_t& s) {
    dim3 threadDims(32, (N > 1)? 32 : 1);
    dim3 blockDims((M + threadDims.x - 1) / threadDims.x, 
                    (N > 1)? ((N + threadDims.y) / threadDims.y) : 1);
    updateParam<<<blockDims, threadDims, 0, s>>>(A, B, lr, M, N);
}

void deviceUpdateStep(HostData& host, DeviceGrads& grads, DeviceNeuralNetwork& dnn, 
                        real learning_rate, CudaStreams& streams) {

    streams.synchronizeAll();
    // Copying from device to host
    checkCudaErrors(hipMemcpyAsync(host.local_dW[0], grads.dW[0], sizeof(real) * grads.H[1] * grads.H[0], 
                  hipMemcpyDeviceToHost, streams.stream[0]));
    checkCudaErrors(hipMemcpyAsync(host.local_db[0], grads.db[0], sizeof(real) * grads.H[1], 
                  hipMemcpyDeviceToHost, streams.stream[2]));
    checkCudaErrors(hipMemcpyAsync(host.local_dW[1], grads.dW[1], sizeof(real) * grads.H[2] * grads.H[1], 
                  hipMemcpyDeviceToHost, streams.stream[1]));
    checkCudaErrors(hipMemcpyAsync(host.local_db[1], grads.db[1], sizeof(real) * grads.H[2], 
                  hipMemcpyDeviceToHost, streams.stream[3]));

    // Performing the MPI Call Here
    MPI_Request reqs[grads.num_layers * 2];
    hipStreamSynchronize(streams.stream[0]);
    MPI_SAFE_CALL(MPI_Iallreduce(host.local_dW[0], host.sum_dW[0], 
                dnn.H[0] * dnn.H[1], MPI_FP, MPI_SUM, MPI_COMM_WORLD, &reqs[0]));

    hipStreamSynchronize(streams.stream[2]);
    MPI_SAFE_CALL(MPI_Iallreduce(host.local_db[0], host.sum_db[0], 
                dnn.H[1], MPI_FP, MPI_SUM, MPI_COMM_WORLD, &reqs[2]));

    hipStreamSynchronize(streams.stream[1]);
    MPI_SAFE_CALL(MPI_Iallreduce(host.local_dW[1], host.sum_dW[1],  
                dnn.H[1] * dnn.H[2], MPI_FP, MPI_SUM, MPI_COMM_WORLD, &reqs[1]));

    hipStreamSynchronize(streams.stream[3]);
    MPI_SAFE_CALL(MPI_Iallreduce(host.local_db[1], host.sum_db[1],
                dnn.H[2], MPI_FP, MPI_SUM, MPI_COMM_WORLD, &reqs[3])); 

    // Coping back to device
    MPI_Wait(&reqs[0], MPI_STATUS_IGNORE);
    checkCudaErrors(hipMemcpyAsync(grads.dW[0], host.sum_dW[0], sizeof(real) * dnn.H[1] * dnn.H[0], 
                      hipMemcpyHostToDevice, streams.stream[0]));
    MPI_Wait(&reqs[2], MPI_STATUS_IGNORE);
    checkCudaErrors(hipMemcpyAsync(grads.db[0], host.sum_db[0], sizeof(real) * dnn.H[1], 
                      hipMemcpyHostToDevice, streams.stream[2]));
    MPI_Wait(&reqs[1], MPI_STATUS_IGNORE);
    checkCudaErrors(hipMemcpyAsync(grads.dW[1], host.sum_dW[1], sizeof(real) * dnn.H[2] * dnn.H[1], 
                      hipMemcpyHostToDevice, streams.stream[1]));
    MPI_Wait(&reqs[3], MPI_STATUS_IGNORE);
    checkCudaErrors(hipMemcpyAsync(grads.db[1], host.sum_db[1], sizeof(real) * dnn.H[2], 
                      hipMemcpyHostToDevice, streams.stream[3]));

    // Performing the actual update
    deviceUpdateParam(grads.dW[0], dnn.W[0], learning_rate, dnn.H[1], dnn.H[0], streams.stream[0]);
    deviceUpdateParam(grads.db[0], dnn.b[0], learning_rate, dnn.H[1], 1, streams.stream[2]); 
    deviceUpdateParam(grads.dW[1], dnn.W[1], learning_rate, dnn.H[2], dnn.H[1], streams.stream[1]);
    deviceUpdateParam(grads.db[1], dnn.b[1], learning_rate, dnn.H[2], 1, streams.stream[3]); 
}

/*------------------ HELPER FUNCTIONS ---------------------*/

void setToZero(real*& ptr, int size) {
    checkCudaErrors(hipMemset(ptr, 0, sizeof(real) * size));
}

void deviceCleanUp(real* ptr) { 
    checkCudaErrors(hipFree(ptr)); 
}

void deviceMalloc(real*& ptr, int size) {
  checkCudaErrors(hipMalloc(&ptr, sizeof(real) * size));
}

real* deviceToDeviceCopy(real* orig, int size) {
  real* ptr = nullptr;
  checkCudaErrors(hipMalloc(&ptr, sizeof(real) * size));
  checkCudaErrors(hipMemcpy(ptr, orig, sizeof(real) * size, hipMemcpyDeviceToDevice));
  return ptr;
}